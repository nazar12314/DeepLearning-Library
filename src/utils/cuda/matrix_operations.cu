#include "hip/hip_runtime.h"
#include "utils/cuda/matrix_operations.cuh"

__global__ void matrixMulKernel(const double* A, const double* B, double* C, int rowsA, int colsA, int colsB, double batches) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0;
        for (int k = 0; k < colsA; ++k) {
            sum += A[row * colsA + k] * B[col * colsA + k];
        }
        C[row * colsB + col] = sum / batches;
    }
}

void matrix_operations::cuda_dense_backward(const double *A, const double *B, double *C_gpu, int rowsA, int colsA, int colsB, double batches) {
    double *A_dev, *B_dev, *C_dev;
    hipMalloc(&A_dev, rowsA * colsA * sizeof(double));
    hipMalloc(&B_dev, colsA * colsB * sizeof(double));
    hipMalloc(&C_dev, rowsA * colsB * sizeof(double));
    hipMemcpy(A_dev, A, rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B, colsA * colsB * sizeof(double), hipMemcpyHostToDevice);

    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dim_grid((colsB + dim_block.x - 1) / dim_block.x, (rowsA + dim_block.y - 1) / dim_block.y);

    matrixMulKernel<<<dim_grid, dim_block>>>(A_dev, B_dev, C_dev, rowsA, colsA, colsB, batches);

    hipMemcpy(C_gpu, C_dev, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
}

__global__ void forward_kernel(const double* A, const double* B, double* C, int rowsA, int colsA, int colsB, const double* biases) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0;
        for (int k = 0; k < colsA; ++k) {
            sum += A[row * colsA + k] * B[col * colsA + k];
        }
        C[row * colsB + col] = sum + biases[col];
    }
}

void matrix_operations::cuda_dense_forward(double *A, double *B, double *C_gpu, int rowsA, int colsA, int colsB, double* biases) {
    double *A_dev, *B_dev, *C_dev, *biases_dev;
    hipMalloc(&A_dev, rowsA * colsA * sizeof(double));
    hipMalloc(&B_dev, colsA * colsB * sizeof(double));
    hipMalloc(&C_dev, rowsA * colsB * sizeof(double));
    hipMalloc(&biases_dev, colsB * sizeof(double));

    hipMemcpy(A_dev, A, rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B, colsA * colsB * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(biases_dev, biases, colsB * sizeof(double), hipMemcpyHostToDevice);

    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dim_grid((colsB + dim_block.x - 1) / dim_block.x, (rowsA + dim_block.y - 1) / dim_block.y);

    forward_kernel<<<dim_grid, dim_block>>>(A_dev, B_dev, C_dev, rowsA, colsA, colsB, biases_dev);

    hipMemcpy(C_gpu, C_dev, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    hipFree(biases_dev);
}